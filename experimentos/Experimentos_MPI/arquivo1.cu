#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "arquivo1.h"
#include "comm/comm.h"




extern "C" void funcao1(){

	FILE *arquivo;
	int N;
	arquivo = fopen("tamanho_vetor","r");

	fscanf(arquivo,"%d",&N);
	//int N =500;
	fclose(arquivo);
	//printf("Funcao 1 \n");
	int *buffer,*d_buffer ;
	int i;

	//clock_t start,finish;
    	//double totaltime;

	dim3 grid, block;
	block.x = 1024;
	grid.x = (N + block.x - 1) / block.x;


	buffer = (int*) malloc(sizeof(int)*N);

	hipMalloc(&d_buffer,sizeof(int)*N);

	for(i=0;i<N;i++){
		buffer[i] = i+1;
		//printf("%d\t",buffer[i]);
		
	}
	//printf("\n");


	hipMemcpy(d_buffer,buffer,N*sizeof(int),hipMemcpyHostToDevice);
	int j;
	//float mean = 0.0f;
	//for(i=0;i<11;i++){
		//start = clock();
		
		//sendMessage("funcao1","funcao3", INT, (void*)d_buffer, N*N);
		//finish = clock();
		//totaltime=(double)(finish-start)/CLOCKS_PER_SEC; 
		//if(i>0) mean+=totaltime;
		//printf("Tempo iteração: %d Tempo: %f\n",i,totaltime);
		//for(j=0;j<80000;j++){
			//calculate<<<grid,block>>>(d_buffer);
			//compute_0(buffer,N*N);
		//}

		sendMessage("funcao1","funcao2", INT, (void*)d_buffer, N);
		//sendMessage("funcao1","funcao2", INT, (void*)buffer, N*N);
	//}
	//printf("Média final: %f\n",mean/10);

	//printf("Mensagem enviando...\n");
	//sendMessage("funcao1","funcao2", INT, (void*)buffer, 10);
	//sendMessage("funcao1","funcao3", INT, (void*)buffer, 10);
	//printf("Mensagem enviada...\n");
	
	
}
