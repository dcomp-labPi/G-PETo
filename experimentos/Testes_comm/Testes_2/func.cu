
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define checkCuda(error) __checkCuda(error, __FILE__, __LINE__)


__global__ void kernel(int *d_buff, int size){

	int ix = blockIdx.x*blockDim.x + threadIdx.x;

	d_buff[ix] = ix+1;

}


extern "C" {

	inline void __checkCuda(hipError_t error, const char *file, const int line){
	
		if (error != hipSuccess){
			printf("checkCuda error at %s:%i: %s\n", file, line, hipGetErrorString(hipGetLastError()));
			exit(-1);
		}
	
		return;
	}

	void callKernel(int *d_buff,int size){

			dim3 grid, block;
                	block.x = 1024;
                        grid.x = (size + block.x - 1) / block.x;


                        kernel<<<block,grid>>>(d_buff,size);

                        //int *h2_buff;
                        //int i;
                        //h2_buff = (int*)malloc(size*sizeof(int));
                        //checkCuda(cudaMemcpy(h2_buff, d_buff,sizeof(int)*size,cudaMemcpyDeviceToHost));
                        //for(i=0;i<size;i++){
                        //        printf("%d\t",h2_buff[i]);
                       // }
                       // printf("\n");

	}


	void init(int **d_buff, int **d_rank,int *rank,int size){
		
		checkCuda(hipMalloc((void**)d_buff,sizeof(int)*size));
		//checkCuda(cudaMalloc((void**)d_rank,sizeof(int)));
		//checkCuda(cudaMemcpy(*d_rank, rank,sizeof(int),cudaMemcpyHostToDevice));

	}

	void MPI_standard(int *h_buff,int *d_buff,int rank, int size){

		if(rank==0){
			checkCuda(hipMemcpy(h_buff, d_buff,sizeof(int)*size,hipMemcpyDeviceToHost));	
		}else{
			checkCuda(hipMemcpy(d_buff, h_buff,sizeof(int)*size,hipMemcpyHostToDevice));


			dim3 grid, block;
	                block.x = 1024;
        	        grid.x = (size + block.x - 1) / block.x;

			
			kernel<<<block,grid>>>(d_buff,size);

                	//int *h2_buff;
			//int i;
                	//h2_buff = (int*)malloc(size*sizeof(int));
                	//checkCuda(cudaMemcpy(h2_buff, d_buff,sizeof(int)*size,cudaMemcpyDeviceToHost));
                	//for(i=0;i<size;i++){
                        //	printf("%d\t",h2_buff[i]);
                	//}
                	//printf("\n");
		}

	}

	void transfer_intra_P2P(int n_buffer){

		int gpu1 = 0;
		int gpu2 = 1;
		int *d_buffer;
		int *d2_buffer;
		int i;

		//int nDevices;

  		//cudaGetDeviceCount(&nDevices);

		//printf("Number of Devices: %d\n",nDevices);


		dim3 grid, block;
		block.x = 1024;
		grid.x = (n_buffer + block.x - 1) / block.x;
		
		//printf("Antes de criar stream_0\n");
		checkCuda(hipSetDevice(gpu1));
		hipStream_t stream_0;
		checkCuda(hipStreamCreate(&stream_0));
		//printf("Antes de alocar d_buffer\n");
		checkCuda(hipMalloc(&d_buffer,sizeof(int)*n_buffer));

		checkCuda(hipSetDevice(gpu2));
		//printf("Antes de alocar d2_buffer\n");
		checkCuda(hipMalloc(&d2_buffer,sizeof(int)*n_buffer));
		//printf("Antes de entrar no for que envia os pacotes\n");
		for(i=0;i<1;i++){
			//printf("Entrei no for i: %d \n",i);
			checkCuda(hipMemcpyPeerAsync(d2_buffer,gpu2,d_buffer,gpu1,n_buffer*sizeof(int),stream_0));
			hipDeviceSynchronize();
		
			kernel<<<block,grid>>>(d2_buffer,n_buffer);
			
		}
		
		checkCuda(hipFree(d2_buffer));
		checkCuda(hipSetDevice(gpu1));
                checkCuda(hipFree(d_buffer));
           
		
		//int *h2_buff;
		//h2_buff = (int*)malloc(n_buffer*sizeof(int));
		//checkCuda(cudaMemcpy(h2_buff, d2_buffer,sizeof(int)*n_buffer,cudaMemcpyDeviceToHost));
		//for(i=0;i<n_buffer;i++){
		//	printf("%d\t",h2_buff[i]);
		//}
		//printf("\n");

	}

	void transfer_intra_standard(int n_buffer){

		int gpu1 = 0;
		int gpu2 = 1;
		int *d_buffer;
		int *d2_buffer;
		int *buffer;
		int i;
	
		dim3 grid, block;
                block.x = 1024;
                grid.x = (n_buffer + block.x - 1) / block.x;



		buffer = (int*) malloc(sizeof(int)*n_buffer);


		checkCuda(hipSetDevice(gpu1));
		checkCuda(hipMalloc(&d_buffer,sizeof(int)*n_buffer));
		//checkCuda(cudaMemcpy(buffer,d_buffer,n_buffer*sizeof(int),cudaMemcpyDeviceToHost));

		checkCuda(hipSetDevice(gpu2));
		checkCuda(hipMalloc(&d2_buffer,sizeof(int)*n_buffer));
		//checkCuda(cudaMemcpy(d2_buffer,buffer,n_buffer*sizeof(int),cudaMemcpyHostToDevice));

		for(i=0;i<1;i++){
			checkCuda(hipSetDevice(gpu1));
			checkCuda(hipMemcpy(buffer,d_buffer,n_buffer*sizeof(int),hipMemcpyDeviceToHost));
			checkCuda(hipSetDevice(gpu2));
			checkCuda(hipMemcpy(d2_buffer,buffer,n_buffer*sizeof(int),hipMemcpyHostToDevice));
			kernel<<<block,grid>>>(d2_buffer,n_buffer);
			hipDeviceSynchronize();
		}
		checkCuda(hipSetDevice(gpu1));
		checkCuda(hipFree(d_buffer));
		checkCuda(hipSetDevice(gpu2));
		checkCuda(hipFree(d2_buffer));

                //int *h2_buff;
		//int i;
                //h2_buff = (int*)malloc(n_buffer*sizeof(int));
                //checkCuda(cudaMemcpy(h2_buff, d2_buffer,sizeof(int)*n_buffer,cudaMemcpyDeviceToHost));
                //for(i=0;i<n_buffer;i++){
                //        printf("%d\t",h2_buff[i]);
                //}
                //printf("\n");

	}

	void setDevice(int device){
		hipSetDevice(device);
	}

	void getResult(int *d_buff, int *h_buff,int size){

		checkCuda(hipMemcpy(h_buff,d_buff,size*sizeof(int),hipMemcpyDeviceToHost));

	}

	void clean(int **d_buff, int **d_rank){
		checkCuda(hipFree(*d_buff));
		checkCuda(hipFree(*d_rank));
	}


}
