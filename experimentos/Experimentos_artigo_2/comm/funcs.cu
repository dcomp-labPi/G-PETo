#include <stdio.h>
#include <stdlib.h>

#include "funcs.h"


//extern "C"{

	void setDevice(int device){
		hipSetDevice(device);
		
	}

	void cpyToCPU(void *buffer,void *d_buffer,int size,Type tipo){
		if(tipo == INT){
			int *d_buffer_local,*buffer_local;
			d_buffer_local = (int*)d_buffer;
			buffer_local = (int*)buffer;
			hipMemcpy(buffer_local,d_buffer_local,sizeof(int)*size,hipMemcpyDeviceToHost);
		}
		else if(tipo == FLOAT){
			float *d_buffer_local_2,*buffer_local_2;
                        d_buffer_local_2 = (float*)d_buffer;
                        buffer_local_2 = (float*)buffer;
			hipMemcpy(buffer_local_2,d_buffer_local_2,sizeof(float)*size,hipMemcpyDeviceToHost);
		}
	}

	void cpyToGPU(void *buffer, void *d_buffer, int size, Type tipo){
		if(tipo == INT){
			int *d_buffer_local,*buffer_local;
                        d_buffer_local = (int*)d_buffer;
                        buffer_local = (int*)buffer;
			
			hipMemcpy(d_buffer_local,buffer_local,size*sizeof(int),hipMemcpyHostToDevice);
		}
		else if(tipo == FLOAT){
			float *d_buffer_local_2,*buffer_local_2;
                        d_buffer_local_2 = (float*)d_buffer;
                        buffer_local_2 = (float*)buffer;
			hipMemcpy(d_buffer_local_2,buffer_local_2,size*sizeof(float),hipMemcpyHostToDevice);
		}
	}

//}
