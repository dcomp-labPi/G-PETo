#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "arquivo2.h"
#include "comm/comm.h"


__global__ void compute1(int *d_buffer){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;

	d_buffer[ix] = d_buffer[ix]*2;




}


extern "C" void funcao2(){


	//printf("Funcao 2 \n");
	int N=500;
	int *buffer,*d_buffer;
	int i,j;//,sum;


	dim3 grid, block;
	block.x = 1024;
	grid.x = (N + block.x - 1) / block.x;


	//buffer = (int*) malloc(sizeof(int)*N*N);
	hipMalloc(&d_buffer,sizeof(int)*N*N);
	//sum = 0;
	for(i=0;i<11;i++){
		receiveMessage("funcao2","funcao1", INT, (void*)d_buffer, N*N);
		//receiveMessage("funcao2","funcao1", INT, (void*)buffer, N*N);

		for(j=0;j<80000;j++){
			compute1<<<grid,block>>>(d_buffer);
			//compute_1(buffer, N*N);
		}
		//hipMemcpy(buffer,d_buffer,N*N*sizeof(int),hipMemcpyDeviceToHost);
		
		//for(j=0;j<N*N;j++){
		//	printf("%d\t",buffer[j]);
		//	sum = sum + buffer[j];
		//}
		//printf("\n");
		//printf("\tFuncao 2 -> Soma Parcial i=%d : %d\n",i,sum);
		sendMessage("funcao2","funcao3", INT, (void*)d_buffer, N*N);
		//sendMessage("funcao2","funcao3", INT, (void*)buffer, N*N);
	}

	//printf("Soma Função 2: %d\n",sum);

	//printf("Recebendo Mensagem...\n");
	//receiveMessage("funcao2","funcao1", INT, (void*)buffer, 10);
	//for(i=0;i<10;i++){
	//	printf("Buffer[%d]: %d\n",i,buffer[i]);
	//}
	//printf("Mensagem recebida...\n");
	

}
