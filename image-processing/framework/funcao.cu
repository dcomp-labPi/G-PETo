#include "hip/hip_runtime.h"
#include "funcao.h"

/*===================== KERNEL CUDA ====================================================================*/
__global__ void sepia(RGBA *d_buffer){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    d_buffer[id].r = (d_buffer[id].r * 0.393f) + (d_buffer[id].g * 0.769f) + (d_buffer[id].b * 0.189f);
    d_buffer[id].g = (d_buffer[id].r * 0.349f) + (d_buffer[id].g * 0.686f) + (d_buffer[id].b * 0.168f);
    d_buffer[id].b = (d_buffer[id].r * 0.272f) + (d_buffer[id].g * 0.534f) + (d_buffer[id].b * 0.131f);
}

__global__ void negative(RGBA *d_buffer){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	d_buffer[ix].r = 255 - d_buffer[ix].r;
	d_buffer[ix].g = 255 - d_buffer[ix].g;
	d_buffer[ix].b = 255 - d_buffer[ix].b;
}

__global__ void grayscale(RGBA *d_buffer){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int luminosidade = d_buffer[ix].r + d_buffer[ix].g + d_buffer[ix].b;

	d_buffer[ix].r = luminosidade/3;
	d_buffer[ix].g = luminosidade/3;
	d_buffer[ix].b = luminosidade/3;
}
/*===================== KERNEL CUDA ====================================================================*/

int size = 150*100;
int quantidade = 5000;
char img_name[] = "150*100.png";
/*=============================== Envia ===============================*/
extern "C" void funcaoEnv1(){
	MPI_Datatype mpi_rgba_type = create_mpi_rgba();

	clock_t start,end; start=clock();

	PNG_DATA *arquivo = read_png_file(img_name);
	RGBA *pixels = translate_px_to_vec(arquivo);

	sendMessage("funcaoEnv1","funcaoProc1", MPI_RGBA, pixels, size);
	receiveMessage("funcaoEnv1","funcaoProc1",MPI_RGBA,pixels,size);

	//translate_vec_to_px(pixels,arquivo);
	//write_png_file("saida.png",arquivo);
	end = clock();

	printf("F1: %lf\n",(double)(end - start) / CLOCKS_PER_SEC);
	MPI_Type_free(&mpi_rgba_type);
}
/*============================== Processadores =====================================*/

void processa_vetor_pixel(RGBA *buffer, int size, int device, int op){
	hipSetDevice(device);
	RGBA *d_buffer;
	hipMalloc((void **)&d_buffer,sizeof(RGBA)*size);
	hipMemcpy(d_buffer, buffer,(size*sizeof(RGBA)),hipMemcpyHostToDevice);

	int i;
	for(i=0; i<200; i++){
		if(op==0){
			sepia<<<size/512,512>>>(d_buffer);
		}else if (op == 1){
			negative<<<size/512,512>>>(d_buffer);
		}else if (op == 2){
			grayscale<<<size/512,512>>>(d_buffer);
		}
	}
	hipMemcpy(buffer, d_buffer, (size*sizeof(RGBA)), hipMemcpyDeviceToHost);
	hipFree(d_buffer);
}

extern "C" void funcaoProc1(){
	int i;

	RGBA *buffer = (RGBA*)malloc(sizeof(RGBA)*size);
	//printf("F1 %d\n",quantidade);
	receiveMessage("funcaoProc1","funcaoEnv1",MPI_RGBA,buffer,size);
	processa_vetor_pixel(buffer,size,0,0);
	sendMessage("funcaoProc1","funcaoProc2",MPI_RGBA,buffer,size);

	for( i=0;i < quantidade-1; i++){
		buffer = (RGBA*)malloc(sizeof(RGBA)*size);
		receiveMessage("funcaoProc1","funcaoProc3",MPI_RGBA,buffer,size);
		processa_vetor_pixel(buffer,size,0,0);
		sendMessage("funcaoProc1","funcaoProc2",MPI_RGBA,buffer,size);
	}

	receiveMessage("funcaoProc1","funcaoProc3",MPI_RGBA,buffer,size);
	sendMessage("funcaoProc1","funcaoEnv1",MPI_RGBA,buffer,size);
}

extern "C" void funcaoProc2(){
	int i;
	//printf("F2 %d\n",quantidade);

	for( i=0; i < quantidade; i++){
		RGBA *buffer = (RGBA*)malloc(sizeof(RGBA)*size);
		receiveMessage("funcaoProc2","funcaoProc1",MPI_RGBA,buffer,size);
		processa_vetor_pixel(buffer,size,1,1);
		sendMessage("funcaoProc2","funcaoProc3",MPI_RGBA,buffer,size);
	}
}

extern "C" void funcaoProc3(){
	int i;
	//printf("F3 %d\n",quantidade);
	
	for( i=0;i < quantidade; i++){
		RGBA *buffer = (RGBA*)malloc(sizeof(RGBA)*size);
		receiveMessage("funcaoProc3","funcaoProc2",MPI_RGBA,buffer,size);
		processa_vetor_pixel(buffer,size,0,2);
		sendMessage("funcaoProc3","funcaoProc1",MPI_RGBA,buffer,size);
	}
}
