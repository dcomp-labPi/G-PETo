#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <mpi.h>
#include <png.h>

typedef struct {
    int r;
    int g;
    int b;
    int a;
} RGBA;

typedef struct{
    int size;
    int width, height;
    png_byte color_type;
    png_byte bit_depth;
    png_bytep *row_pointers;
    RGBA* pixels;
}PNG_DATA;

void copy_pixels(RGBA *position, png_bytep px2){
    position->r = px2[0];
    position->g = px2[1];
    position->b = px2[2];
    position->a = px2[3];
}

void copy_positions(RGBA position, png_bytep px2){
    px2[0] = position.r;
    px2[1] = position.g;
    px2[2] = position.b;
    px2[3] = position.a;
}

RGBA* translate_px_to_vec(PNG_DATA* dados) {
    int x,y;
    int height = dados->height;
    int width = dados->width;
    RGBA *pixels = (RGBA*)malloc(sizeof(RGBA)* dados->size);
    for (y = 0; y < height; y++) {
        png_bytep row = dados->row_pointers[y];
        for (x = 0; x < width; x++) {
            png_bytep px = &(row[x * 4]);
            copy_pixels(&(pixels[y*width+x]),px);
        }
    }
    return pixels;
}

void translate_vec_to_px(RGBA* input, PNG_DATA* dados){
    int x,y;
    int height = dados->height;
    int width = dados->width;
    for (y = 0; y < height; y++) {
        png_bytep row = dados->row_pointers[y];
        for (x = 0; x < width; x++) {
            png_bytep px = &(row[x * 4]);
            copy_positions(input[y*width+x],px);
        }
    }
}



/*** Leitura e escrita no png ***/
PNG_DATA* read_png_file(char *filename) {
    int y;
    PNG_DATA *novo = (PNG_DATA*)malloc(sizeof(PNG_DATA));
    FILE *fp = fopen(filename, "rb");

    png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info = png_create_info_struct(png);
    if (!png || !info)
        abort();

    if (setjmp(png_jmpbuf(png)))
        abort();

    png_init_io(png, fp);

    png_read_info(png, info);

    novo->width = png_get_image_width(png, info);
    novo->height = png_get_image_height(png, info);
    novo->size = novo->width*novo->height;
    novo->color_type = png_get_color_type(png, info);
    novo->bit_depth = png_get_bit_depth(png, info);

    if (novo->bit_depth == 16)
        png_set_strip_16(png);

    if (novo->color_type == PNG_COLOR_TYPE_PALETTE)
        png_set_palette_to_rgb(png);

    // PNG_COLOR_TYPE_GRAY_ALPHA is always 8 or 16bit depth.
    if (novo->color_type == PNG_COLOR_TYPE_GRAY && novo->bit_depth < 8)
        png_set_expand_gray_1_2_4_to_8(png);

    if (png_get_valid(png, info, PNG_INFO_tRNS))
        png_set_tRNS_to_alpha(png);

    // These color_type don't have an alpha channel then fill it with 0xff.
    if (novo->color_type == PNG_COLOR_TYPE_RGB || novo->color_type == PNG_COLOR_TYPE_GRAY ||
            novo->color_type == PNG_COLOR_TYPE_PALETTE)
        png_set_filler(png, 0xFF, PNG_FILLER_AFTER);

    if (novo->color_type == PNG_COLOR_TYPE_GRAY ||
            novo->color_type == PNG_COLOR_TYPE_GRAY_ALPHA)
        png_set_gray_to_rgb(png);

    png_read_update_info(png, info);

    novo->row_pointers = (png_bytep *)malloc(sizeof(png_bytep) * novo->height);
    for (y = 0; y < novo->height; y++) {
        novo->row_pointers[y] = (png_byte *)malloc(png_get_rowbytes(png, info));
    }

    png_read_image(png, novo->row_pointers);

    fclose(fp);
    return novo;
}

void write_png_file(char *filename, PNG_DATA* dados) {
    
    int width = dados->width;
    int height = dados->height;

    FILE *fp = fopen(filename, "wb");
    if (!fp)
        abort();

    png_structp png =
            png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png)
        abort();

    png_infop info = png_create_info_struct(png);
    if (!info)
        abort();

    if (setjmp(png_jmpbuf(png)))
        abort();

    png_init_io(png, fp);

    // Output is 8bit depth, RGBA format.
    png_set_IHDR(png, info, width, height, 8, PNG_COLOR_TYPE_RGBA,
                             PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_DEFAULT,
                             PNG_FILTER_TYPE_DEFAULT);
    png_write_info(png, info);

    // To remove the alpha channel for PNG_COLOR_TYPE_RGB format,
    // Use png_set_filler().
    // png_set_filler(png, 0, PNG_FILLER_AFTER);

    png_write_image(png, dados->row_pointers);
    png_write_end(png, NULL);

    /*for (y = 0; y < height; y++) {
        free(dados->row_pointers[y]);
    }*/
    //free(dados->row_pointers);

    fclose(fp);
}

/*=============================================================================================*/

__global__ void grayscale(RGBA *pixels){
    int id = blockIdx.x*blockDim.x + threadIdx.x;   
    int result = 0.21f * pixels[id].r + 0.72f * pixels[id].g + 0.07f * pixels[id].b;
    pixels[id].r = pixels[id].g = pixels[id].b = result;
}
__global__ void negative(RGBA *pixels){
    int id = blockIdx.x*blockDim.x + threadIdx.x;   
    pixels[id].r = 255-pixels[id].r;
    pixels[id].g = 255-pixels[id].g;
    pixels[id].b = 255-pixels[id].b;
}

__global__ void sepia(RGBA *pixels){
    int id = blockIdx.x*blockDim.x + threadIdx.x;   
    pixels[id].r = (pixels[id].r * 0.393f) + (pixels[id].g * 0.769f) + (pixels[id].b * 0.189f);
    pixels[id].g = (pixels[id].r * 0.349f) + (pixels[id].g * 0.686f) + (pixels[id].b * 0.168f);
    pixels[id].b = (pixels[id].r * 0.272f) + (pixels[id].g * 0.534f) + (pixels[id].b * 0.131f);
}

MPI_Datatype create_mpi_rgba(){
	// Criacao do tipo de dado do mpi para o RGBA *
    const int nitens = 4;
    int blocklengths[4] = {1,1,1,1};
    MPI_Datatype types[4] = {MPI_INT,MPI_INT,MPI_INT,MPI_INT};
    MPI_Datatype mpi_rgba_type;
    MPI_Aint offsets[4];
    offsets[0] = offsetof(RGBA,r);
    offsets[1] = offsetof(RGBA,g);
    offsets[2] = offsetof(RGBA,b);
    offsets[3] = offsetof(RGBA,a);
    MPI_Type_create_struct(nitens,blocklengths,offsets,types,&mpi_rgba_type);
    MPI_Type_commit(&mpi_rgba_type);
	return mpi_rgba_type;
    // Termino da criacao do tipo do mpi para o rgba *
}


int size = 150*100;
int testSize = 5000;
int filterSize = 200;

int main(int argc, char *argv[]){    
    
    MPI_Init(&argc, &argv);

    

    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);        
    MPI_Datatype mpi_rgba_type = create_mpi_rgba();              
    clock_t start,end;
    start=clock();    
    if (world_rank == 0) {        
               
        PNG_DATA* imageData = read_png_file("150x100.png");
        RGBA* result = translate_px_to_vec(imageData);
        
        
        int countControle = 0;
        //(countSend<testSize || countReceived<testSize) && 
        //printf("ProcID %d vai enviar imagem%d para %d (%d).\n",world_rank,0,1,world_size);                
        MPI_Send(result,size,mpi_rgba_type,1,0,MPI_COMM_WORLD);
        //printf("ProcID %d enviou imagem%d para %d (%d).\n",world_rank,0,1,world_size);                
        
        MPI_Recv(result, size,mpi_rgba_type,3, 10, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        
        /*translate_vec_to_px(result,imageData);            
        char test[19];
        s//printf(test,"output/image.png");
        write_png_file(test,imageData);
        end = clock();
        double duration = (double)(end - start) / CLOCKS_PER_SEC;
        //printf("EXECUTION_TIME = %f\n",duration);    */    
        
        MPI_Type_free(&mpi_rgba_type);
        MPI_Finalize();
        
        
    } else if(world_rank == 1){        
        MPI_Status status;     
        RGBA* result = (RGBA*) malloc(sizeof(RGBA)*size);   
        //printf("ProcID %d vai receber imagem%d de %d (%d).\n",world_rank,0,0,world_size);                     
        MPI_Recv(result, size,mpi_rgba_type,0, 0, MPI_COMM_WORLD, &status);
        //printf("ProcID %d vai recebeu imagem%d de %d (%d).\n",world_rank,0,0,world_size);                     
        for(int countControle = 0;countControle<testSize;countControle++){                        
            hipSetDevice(0);
            RGBA* d_result;
            
            hipMalloc((void **) &d_result,(size*sizeof(RGBA)));
            hipMemcpy(d_result,result,(size*sizeof(RGBA)),hipMemcpyHostToDevice);
            for(int i=0;i<filterSize;i++){ //divido por 2 pois cada repetição executa 2 filtros                    
                sepia<<<size/512,512>>>(d_result);                                
            }
            hipMemcpy(result,d_result,(size*sizeof(RGBA)),hipMemcpyDeviceToHost);
            //printf("ProcID %d vai enviar imagem%d para %d (%d).\n",world_rank,0,2,world_size);                     
            MPI_Send(result,size,mpi_rgba_type,2,10,MPI_COMM_WORLD);                            
            //printf("ProcID %d vai enviaou imagem%d para %d (%d).\n",world_rank,0,2,world_size);  
            if(countControle < (testSize-1)){
                RGBA* result = (RGBA*) malloc(sizeof(RGBA)*size);        
                //printf("ProcID %d vai receber imagem%d de %d (%d).\n",world_rank,0,3,world_size);                     
                
                MPI_Recv(result, size,mpi_rgba_type,3, 10, MPI_COMM_WORLD, &status);
                //printf("ProcID %d vai recebeu imagem%d de %d (%d).\n",world_rank,0,3,world_size);                     
            }                   
        }
        MPI_Finalize();
    }else if(world_rank == 2){
        for(int countControle = 0;countControle<testSize;countControle++){                        
            hipSetDevice(1);
            RGBA* result = (RGBA*) malloc(sizeof(RGBA)*size);        
            MPI_Status status; 
            //printf("ProcID %d vai receber imagem%d para %d (%d).\n",world_rank,0,1,world_size);                         
            MPI_Recv(result, size,mpi_rgba_type,1, 10, MPI_COMM_WORLD, &status);            
            RGBA* d_result;
            
            hipMalloc((void **) &d_result,(size*sizeof(RGBA)));
            hipMemcpy(d_result,result,(size*sizeof(RGBA)),hipMemcpyHostToDevice);
            //printf("ProcID %d vai recebeu imagem%d para %d (%d).\n",world_rank,0,1,world_size);                         
            for(int i=0;i<filterSize;i++){ //divido por 2 pois cada repetição executa 2 filtros                    
                negative<<<size/512,512>>>(d_result);                                
                //hipMemcpy(result,d_result,(size*sizeof(RGBA)),hipMemcpyDeviceToHost);                
            }
            hipMemcpy(result,d_result,(size*sizeof(RGBA)),hipMemcpyDeviceToHost);
            
           // MPI_Abort(MPI_COMM_WORLD,MPI_SUCCESS);
            //printf("ProcID %d vai enviar imagem%d para %d (%d).\n",world_rank,0,3,world_size);                                                
            MPI_Send(result,size,mpi_rgba_type,3,10,MPI_COMM_WORLD);     
            //printf("ProcID %d vai enviou imagem%d para %d (%d).\n",world_rank,0,3,world_size);                                                
        }
        MPI_Finalize();

    }else if(world_rank == 3){

        for(int countControle = 0;countControle<testSize;countControle++){                        
            hipSetDevice(0);
        
            RGBA* result = (RGBA*) malloc(sizeof(RGBA)*size);        
            MPI_Status status; 
            RGBA* d_result;    
            //printf("ProcID %d vai receber imagem%d para %d (%d).\n",world_rank,0,2,world_size);                                                    
            MPI_Recv(result, size,mpi_rgba_type,2, 10, MPI_COMM_WORLD, &status);            
            hipMalloc((void **) &d_result,(size*sizeof(RGBA)));
            hipMemcpy(d_result,result,(size*sizeof(RGBA)),hipMemcpyHostToDevice);            
            
            //printf("ProcID %d vai recebeu imagem%d para %d (%d).\n",world_rank,0,2,world_size);                                                    
            for(int i=0;i<filterSize;i++){ //divido por 2 pois cada repetição executa 2 filtros                    
                grayscale<<<size/512,512>>>(d_result);                                
            }
            
            
            if(countControle < (testSize-1)){        
                //printf("ProcID %d vai enviar imagem%d para %d (%d).\n",world_rank,0,1,world_size);    
                hipMemcpy(result,d_result,(size*sizeof(RGBA)),hipMemcpyDeviceToHost);                                                
                MPI_Send(result,size,mpi_rgba_type,1,10,MPI_COMM_WORLD);                            
                //printf("ProcID %d vai enviou imagem%d para %d (%d).\n",world_rank,0,1,world_size);                                                    
            }else{
                //hipMemcpy(result,d_result,(size*sizeof(RGBA)),hipMemcpyDeviceToHost);        
                //MPI_Send(result,size,mpi_rgba_type,0,10,MPI_COMM_WORLD);                                    
            }
        }        
        end = clock();
        double duration = (double)(end - start) / CLOCKS_PER_SEC;
        printf("EXECUTION_TIME = %f\n",duration); 
        MPI_Abort(MPI_COMM_WORLD,MPI_SUCCESS);                
    }
    return 0;
}
