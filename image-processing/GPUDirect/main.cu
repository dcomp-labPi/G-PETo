
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//#include <mpi.h>
#include <png.h>

#define checkCuda(error) __checkCuda(error, __FILE__, __LINE__)

typedef struct {
    int r;
    int g;
    int b;
    int a;
} RGBA;

typedef struct{
    int size;
    int width, height;
    png_byte color_type;
    png_byte bit_depth;
    png_bytep *row_pointers;
    RGBA* pixels;
}PNG_DATA;

void copy_pixels(RGBA *position, png_bytep px2){
    position->r = px2[0];
    position->g = px2[1];
    position->b = px2[2];
    position->a = px2[3];
}

void copy_positions(RGBA position, png_bytep px2){
    px2[0] = position.r;
    px2[1] = position.g;
    px2[2] = position.b;
    px2[3] = position.a;
}

RGBA* translate_px_to_vec(PNG_DATA* dados) {
    int x,y;
    int height = dados->height;
    int width = dados->width;
    RGBA *pixels = (RGBA*)malloc(sizeof(RGBA)* dados->size);
    for (y = 0; y < height; y++) {
        png_bytep row = dados->row_pointers[y];
        for (x = 0; x < width; x++) {
            png_bytep px = &(row[x * 4]);
            copy_pixels(&(pixels[y*width+x]),px);
        }
    }
    return pixels;
}

void translate_vec_to_px(RGBA* input, PNG_DATA* dados){
    int x,y;
    int height = dados->height;
    int width = dados->width;
    for (y = 0; y < height; y++) {
        png_bytep row = dados->row_pointers[y];
        for (x = 0; x < width; x++) {
            png_bytep px = &(row[x * 4]);
            copy_positions(input[y*width+x],px);
        }
    }
}



/*** Leitura e escrita no png ***/
PNG_DATA* read_png_file(char *filename) {
    int y;
    PNG_DATA *novo = (PNG_DATA*)malloc(sizeof(PNG_DATA));
    FILE *fp = fopen(filename, "rb");

    png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info = png_create_info_struct(png);
    if (!png || !info)
        abort();

    if (setjmp(png_jmpbuf(png)))
        abort();

    png_init_io(png, fp);

    png_read_info(png, info);

    novo->width = png_get_image_width(png, info);
    novo->height = png_get_image_height(png, info);
    novo->size = novo->width*novo->height;
    novo->color_type = png_get_color_type(png, info);
    novo->bit_depth = png_get_bit_depth(png, info);

    // Read any color_type into 8bit depth, RGBA format.
    // See http://www.libpng.org/pub/png/libpng-manual.txt

    if (novo->bit_depth == 16)
        png_set_strip_16(png);

    if (novo->color_type == PNG_COLOR_TYPE_PALETTE)
        png_set_palette_to_rgb(png);

    // PNG_COLOR_TYPE_GRAY_ALPHA is always 8 or 16bit depth.
    if (novo->color_type == PNG_COLOR_TYPE_GRAY && novo->bit_depth < 8)
        png_set_expand_gray_1_2_4_to_8(png);

    if (png_get_valid(png, info, PNG_INFO_tRNS))
        png_set_tRNS_to_alpha(png);

    // These color_type don't have an alpha channel then fill it with 0xff.
    if (novo->color_type == PNG_COLOR_TYPE_RGB || novo->color_type == PNG_COLOR_TYPE_GRAY ||
            novo->color_type == PNG_COLOR_TYPE_PALETTE)
        png_set_filler(png, 0xFF, PNG_FILLER_AFTER);

    if (novo->color_type == PNG_COLOR_TYPE_GRAY ||
            novo->color_type == PNG_COLOR_TYPE_GRAY_ALPHA)
        png_set_gray_to_rgb(png);

    png_read_update_info(png, info);

    novo->row_pointers = (png_bytep *)malloc(sizeof(png_bytep) * novo->height);
    for (y = 0; y < novo->height; y++) {
        novo->row_pointers[y] = (png_byte *)malloc(png_get_rowbytes(png, info));
    }

    png_read_image(png, novo->row_pointers);

    fclose(fp);
    return novo;
}

void write_png_file(char *filename, PNG_DATA* dados) {
    
    int width = dados->width;
    int height = dados->height;

    FILE *fp = fopen(filename, "wb");
    if (!fp)
        abort();

    png_structp png =
            png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png)
        abort();

    png_infop info = png_create_info_struct(png);
    if (!info)
        abort();

    if (setjmp(png_jmpbuf(png)))
        abort();

    png_init_io(png, fp);

    // Output is 8bit depth, RGBA format.
    png_set_IHDR(png, info, width, height, 8, PNG_COLOR_TYPE_RGBA,
                             PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_DEFAULT,
                             PNG_FILTER_TYPE_DEFAULT);
    png_write_info(png, info);

    // To remove the alpha channel for PNG_COLOR_TYPE_RGB format,
    // Use png_set_filler().
    // png_set_filler(png, 0, PNG_FILLER_AFTER);

    png_write_image(png, dados->row_pointers);
    png_write_end(png, NULL);

    /*for (y = 0; y < height; y++) {
        free(dados->row_pointers[y]);
    }*/
    //free(dados->row_pointers);

    fclose(fp);
}

/*=============================================================================================*/

__global__ void grayscale(RGBA *pixels){
    int id = blockIdx.x*blockDim.x + threadIdx.x;   
    int result = 0.21f * pixels[id].r + 0.72f * pixels[id].g + 0.07f * pixels[id].b;
    pixels[id].r = pixels[id].g = pixels[id].b = result;
}
__global__ void negative(RGBA *pixels){
    int id = blockIdx.x*blockDim.x + threadIdx.x;   
    pixels[id].r = 255-pixels[id].r;
    pixels[id].g = 255-pixels[id].g;
    pixels[id].b = 255-pixels[id].b;
}

__global__ void sepia(RGBA *pixels){
    int id = blockIdx.x*blockDim.x + threadIdx.x;   
    pixels[id].r = (pixels[id].r * 0.393f) + (pixels[id].g * 0.769f) + (pixels[id].b * 0.189f);
    pixels[id].g = (pixels[id].r * 0.349f) + (pixels[id].g * 0.686f) + (pixels[id].b * 0.168f);
    pixels[id].b = (pixels[id].r * 0.272f) + (pixels[id].g * 0.534f) + (pixels[id].b * 0.131f);
}

int size=150*100;
int testSize = 1000;
int processar = 400;

int main(int argc, char *argv[]){  
	int cont = 0;  
	int gpu1 = 0;
	int gpu2 = 1;
	RGBA *d1_buffer;
	RGBA *d2_buffer;
	RGBA* result;

	clock_t start,end;
	start=clock();    

	char test[] = "150x100.png";
	PNG_DATA* imageData = read_png_file(test);

	hipSetDevice(gpu1);
	hipMalloc(&d1_buffer, (size*sizeof(RGBA)));
	hipSetDevice(gpu2);
	hipMalloc(&d2_buffer, (size*sizeof(RGBA)));
	hipMalloc(&result, (size*sizeof(RGBA)));

	while (cont < testSize) {
		hipSetDevice(gpu1);  

		//cudaStream_t stream_0;
		//cudaStreamCreate(&stream_0);

		//cudaMalloc(&d1_buffer, (size*sizeof(RGBA)));

		//RGBA* result = translate_px_to_vec(imageData);
		result = translate_px_to_vec(imageData);
		
		//cudaSetDevice(gpu2);
		//cudaMalloc(&d2_buffer, (size*sizeof(RGBA)));

		hipMemcpyPeer(d2_buffer, gpu2, result, gpu1, (size*sizeof(RGBA)));
		hipDeviceSynchronize();
		
		hipSetDevice(gpu2);
		for (int i = 0; i < processar/2; i++){
			
			sepia<<<size/512,512>>>(d2_buffer);
			
			hipMemcpyPeer(d1_buffer, gpu1, d2_buffer, gpu2, (size*sizeof(RGBA)));
			hipDeviceSynchronize();
			
			hipSetDevice(gpu1); 
			negative<<<size/512,512>>>(d1_buffer);
			
			hipMemcpyPeer(d2_buffer, gpu2, d1_buffer, gpu1, (size*sizeof(RGBA)));
			hipDeviceSynchronize();
			
			hipSetDevice(gpu2);
		}
		
		hipMemcpy(result,d2_buffer,(size*sizeof(RGBA)),hipMemcpyDeviceToHost);
		translate_vec_to_px(result,imageData); 

		cont++;

		char test[16];
		sprintf(test,"finalteste1.png");
		write_png_file(test,imageData);
	}


	hipFree(d1_buffer);
	hipSetDevice(gpu2);
	hipFree(d2_buffer);
	hipFree(result);
	

    end = clock();
    double duration = (double)(end - start) / CLOCKS_PER_SEC;
    printf("EXECUTION_TIME = %f\n",duration);



}
